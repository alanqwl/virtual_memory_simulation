﻿/* /bonus/virtual_memory.cu */

#include "virtual_memory.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ int find_LRU(VirtualMemory *vm);

__device__ void init_invert_page_table(VirtualMemory *vm) {

  for (int i = 0; i < vm->PAGE_ENTRIES; i++) {
    vm->invert_page_table[i] = 0x80000000 + vm->thread_id; // invalid := MSB is 1.
    vm->invert_page_table[i + vm->PAGE_ENTRIES] = i;

    // record the time stamp of each page which would be useful in LRU algorithm
    vm->invert_page_table[i + 2 * vm->PAGE_ENTRIES] = i;  
  }
  vm->invert_page_table[3 * vm->PAGE_ENTRIES] = vm->PAGE_ENTRIES - 1;  // current time stamp
  vm->invert_page_table[3 * vm->PAGE_ENTRIES + 1] = 0;  // LRU entry index
}

__device__ void init_sw_table(VirtualMemory *vm) {

  /* In total 4096(128 kb / 32b = 4096) entries in the table, each connects the secondary addr to the physical addr. */
  for (int i = 0; i < (vm->STORAGE_SIZE / vm->PAGESIZE); i++){
    vm->sw_table[i] = 6000;
  }
}

__device__ void vm_init(VirtualMemory *vm, uchar *buffer, uchar *storage,
                        u32 *invert_page_table, u32 *swap_table, int *pagefault_num_ptr, int threadID, 
                        int PAGESIZE, int INVERT_PAGE_TABLE_SIZE, 
                        int PHYSICAL_MEM_SIZE, int STORAGE_SIZE, 
                        int PAGE_ENTRIES) {
  // init variables
  vm->buffer = buffer;
  vm->storage = storage;
  vm->invert_page_table = invert_page_table;
  vm->sw_table = swap_table;
  vm->pagefault_num_ptr = pagefault_num_ptr;

  // init constants
  vm->thread_id = threadID;
  vm->PAGESIZE = PAGESIZE;
  vm->INVERT_PAGE_TABLE_SIZE = INVERT_PAGE_TABLE_SIZE;
  vm->PHYSICAL_MEM_SIZE = PHYSICAL_MEM_SIZE;
  vm->STORAGE_SIZE = STORAGE_SIZE;
  vm->PAGE_ENTRIES = PAGE_ENTRIES;

  *(vm->pagefault_num_ptr) = 0;

  // before first vm_write or vm_read
  if (vm->thread_id == 0) {
    init_invert_page_table(vm);
    init_sw_table(vm);
  }
}

/* find the minimum value of time stamp which indicates the least recently used entry */
__device__ int find_LRU(VirtualMemory *vm) {
  int min_stamp = vm->invert_page_table[2 * vm->PAGE_ENTRIES];
  int result = 0;
  for (int i = 1; i < vm->PAGE_ENTRIES; i++) {
    if (vm->invert_page_table[2 * vm->PAGE_ENTRIES + i] < min_stamp) {
      min_stamp = vm->invert_page_table[2 * vm->PAGE_ENTRIES + i];
      result = i;
    }
  }
  return result;
}

__device__ uchar vm_read(VirtualMemory *vm, u32 addr) {
  /* Complate vm_read function to read single element from data buffer */
  uchar output = 0;
  u32 page_num = addr / vm->PAGESIZE;   // page number in range [0, 160 * 32 = 5120]
  u32 frame_num;
  u32 physical_address;
  
  // iterate through 1024(10^15 / 10^5 = 1024) page entries.
  for (int i = 0; i < vm->PAGE_ENTRIES; i++) {

    /* page hit */
    if (page_num == vm->invert_page_table[i + vm->PAGE_ENTRIES]) {
      // check the valid bit
      if ((vm->invert_page_table[i] & 0x80000000) != 0x80000000) {
        frame_num = i;    // frame number would be i instead of page_num
        physical_address = (frame_num * vm->PAGESIZE) + (addr & 0x0000001f);  // offset = addr & 0x0000001f(last 5 bits)
        output = vm->buffer[physical_address];
        
        // update the time stamp
        vm->invert_page_table[3 * vm->PAGE_ENTRIES] += 1;
        vm->invert_page_table[frame_num + 2 * vm->PAGE_ENTRIES] = vm->invert_page_table[3 * vm->PAGE_ENTRIES];
        if (vm->invert_page_table[3 * vm->PAGE_ENTRIES + 1] == i) {
          vm->invert_page_table[3 * vm->PAGE_ENTRIES + 1] = find_LRU(vm);
        }

        return output;
      }else{
        /* The valid bit is invalid, load the page from the secondary memory to the physical memory */
        *(vm->pagefault_num_ptr) += 1;
        int secondary_idx1;
        int tr;

        // find the data in the secondary memory according to the page number
        for (tr = 0; tr < (vm->STORAGE_SIZE / vm->PAGESIZE); tr++) {
          if (vm->sw_table[tr] == page_num) {
            secondary_idx1 = tr;
            break;
          }
        }

        // load one page of data from the secondary memory to the physical memory
        for (int k = 0; k < 32; k++) {
          vm->buffer[i * (vm->PAGESIZE) + k] = vm->storage[secondary_idx1 * (vm->PAGESIZE) + k];
        }
        vm->sw_table[secondary_idx1] = 6000;    // update swap table
        output = vm->buffer[(i * vm->PAGESIZE) + (addr & 0x0000001f)];

        vm->invert_page_table[i] &= 0x7fffffff;    // change the invalid bit to valid.

        /* update the time stamp of the page table*/
        vm->invert_page_table[i + 2 * vm->PAGE_ENTRIES] = vm->invert_page_table[3 * vm->PAGE_ENTRIES] + 1;    
        vm->invert_page_table[3 * vm->PAGE_ENTRIES] += 1;
        if (vm->invert_page_table[3 * vm->PAGE_ENTRIES + 1] == i) {
          vm->invert_page_table[3 * vm->PAGE_ENTRIES + 1] = find_LRU(vm);
        }
        return output;
      }
    } 
  }
  /* The page is not in the table: do the swapping */
  *(vm->pagefault_num_ptr) += 1; 
  int swap_idx = vm->invert_page_table[3 * vm->PAGE_ENTRIES + 1];   // select the swap frame index (current)

  /* swap out */
  int tr2, tr3;


  for (tr2 = 0; tr2 < (vm->STORAGE_SIZE / vm->PAGESIZE); tr2++) {
    if (vm->sw_table[tr2] == 6000) {     // find a empty position in the disk
      vm->sw_table[tr2] = vm->invert_page_table[swap_idx + vm->PAGE_ENTRIES];    // update the swap table
      for (tr3 = 0; tr3 < 32; tr3++) { // load the data to the disk
        vm->storage[tr2 * vm->PAGESIZE + tr3] = vm->buffer[swap_idx * vm->PAGESIZE + tr3];
      }
      break;
    }
  }

  /* swap in */
  for (tr3 = 0; tr3 < (vm->STORAGE_SIZE / vm->PAGESIZE); tr3++) {
    if (vm->sw_table[tr3] == page_num) {
      for (tr2 = 0; tr2 < 32; tr2++) {  // load the data to the physical memory
        vm->buffer[swap_idx * vm->PAGESIZE + tr2] = vm->storage[tr3 * vm->PAGESIZE + tr2];
      }
      output = vm->buffer[(swap_idx * vm->PAGESIZE) + (addr & 0x0000001f)];
      vm->invert_page_table[swap_idx + vm->PAGE_ENTRIES] = page_num;  // update page table
      vm->sw_table[tr3] = 6000;
      break;  
    }  
  }

  /* update LRU info */
  vm->invert_page_table[3 * vm->PAGE_ENTRIES] += 1;
  vm->invert_page_table[swap_idx + 2 * vm->PAGE_ENTRIES] = vm->invert_page_table[3 * vm->PAGE_ENTRIES];
  vm->invert_page_table[3 * vm->PAGE_ENTRIES + 1] = find_LRU(vm);

  return output; //TODO
}

__device__ void vm_write(VirtualMemory *vm, u32 addr, uchar value) {
  /* Complete vm_write function to write value into data buffer */
  u32 page_num = addr / vm->PAGESIZE;
  u32 frame_num;
  
  for (int i = 0; i < vm->PAGE_ENTRIES; i++) {

    /* page hit */
    if (page_num == vm->invert_page_table[i + vm->PAGE_ENTRIES]) {
      
      if (!((vm->invert_page_table[i]) >> 31)) {
        frame_num = i;
        vm->buffer[frame_num * vm->PAGESIZE + (addr % vm->PAGESIZE)] = value;

        /* maintain the LRU info */
        vm->invert_page_table[3 * vm->PAGE_ENTRIES] += 1;
        vm->invert_page_table[frame_num + 2 * vm->PAGE_ENTRIES] = vm->invert_page_table[3 * vm->PAGE_ENTRIES];
        if (vm->invert_page_table[3 * vm->PAGE_ENTRIES + 1] == frame_num){
          vm->invert_page_table[3 * vm->PAGE_ENTRIES + 1] = find_LRU(vm);
        }
        return;
      }
      else {
        /* valid bit is set as invalid: load from disk to physical memory */
        *(vm->pagefault_num_ptr) += 1;
        int tr, k;
        int secondary_idx;
        int flag = 0;
        for (tr = 0; tr < (vm->STORAGE_SIZE / vm->PAGESIZE); tr++){
          if (page_num == vm->sw_table[tr]) {
            secondary_idx = tr;
            flag = 1;
            break;
          }
        }

        if (!flag) {
          vm->buffer[i * vm->PAGESIZE + (addr % vm->PAGESIZE)] = value;
        }else{
          // load the data from the disk to the physical memory
          for (k = 0; k < 32; k++){
            vm->buffer[i * (vm->PAGESIZE) + k] = vm->storage[secondary_idx * vm->PAGESIZE + k];
          }
          vm->sw_table[secondary_idx] = 6000;   // update swap table 
          vm->buffer[i * vm->PAGESIZE + (addr % vm->PAGESIZE)] = value;
        }

        vm->invert_page_table[i] = 0x00000000;  // set the invalid bit to valid

        /* maintain the LRU info */
        vm->invert_page_table[3 * vm->PAGE_ENTRIES] += 1;
        vm->invert_page_table[i + 2 * vm->PAGE_ENTRIES] = vm->invert_page_table[3 * vm->PAGE_ENTRIES];
        if (vm->invert_page_table[3 * vm->PAGE_ENTRIES + 1] == i) {
          vm->invert_page_table[3 * vm->PAGE_ENTRIES + 1] = find_LRU(vm);
        }
        return;  
      }
    }
  }

  // check whether the address is an illegel one
  if (addr >= 163840) {
    printf("illegel address");
    return;
  }
  *(vm->pagefault_num_ptr) += 1;
  /* check whether there is empty frame */
  int tr2, tr3;
  int empty_frame;
  for (tr2 = 0; tr2 < vm->PAGE_ENTRIES; tr2++) {
    if ((vm->invert_page_table[tr2] & 0x80000000) == 0x80000000) {
      empty_frame = tr2;
      
      vm->invert_page_table[empty_frame + vm->PAGE_ENTRIES] = page_num;
      vm->invert_page_table[empty_frame] = 0x00000000;  // change the invalid bit to valid
      vm->buffer[(empty_frame * vm->PAGESIZE) + (addr % vm->PAGESIZE)] = value;

      // update LRU info
      vm->invert_page_table[3 * vm->PAGE_ENTRIES] += 1;
      vm->invert_page_table[empty_frame + 2 * vm->PAGE_ENTRIES] = vm->invert_page_table[3 * vm->PAGE_ENTRIES];
      if (vm->invert_page_table[3 * vm->PAGE_ENTRIES + 1] == empty_frame) {
        vm->invert_page_table[3 * vm->PAGE_ENTRIES + 1] = find_LRU(vm);
      }
      return;
    }
  }

  /* no empty frame: swapping */

  int swap_idx = vm->invert_page_table[3 * vm->PAGE_ENTRIES + 1];
  int swap_page = vm->invert_page_table[swap_idx + vm->PAGE_ENTRIES];  
  /* swap out */
  int flag2 = 0;
  for (tr2 = 0; tr2 < (vm->STORAGE_SIZE / vm->PAGESIZE); tr2++) {
    if (vm->sw_table[tr2] == 6000) {
      vm->sw_table[tr2] = swap_page;
      flag2 = 1;
      for (tr3 = 0; tr3 < 32; tr3++) {
        vm->storage[tr2 * vm->PAGESIZE + tr3] = vm->buffer[swap_idx * vm->PAGESIZE + tr3];
      }
      break; 
    }
  }

  /* swap in */
  int flag3 = 0;
  int secondary_idx2;
  for (tr3 = 0; tr3 < (vm->STORAGE_SIZE / vm->PAGESIZE); tr3++) {
    if (vm->sw_table[tr3] == page_num) {
      flag3 = 1;
      secondary_idx2 = tr3;
      for (tr2 = 0; tr2 < 32; tr2++) {
        vm->buffer[swap_idx * vm->PAGESIZE + tr2] = vm->storage[tr3 * vm->PAGESIZE + tr2];
      }
      vm->invert_page_table[swap_idx + vm->PAGE_ENTRIES] = page_num;
      vm->sw_table[tr3] = 6000;
      break;
    }
  }
  
  if (flag2 == 0 && flag3 == 1) {   // both the frame and the secondary memory are full
    vm->sw_table[secondary_idx2] = swap_page;
    for (tr2 = 0; tr2 < 32; tr2++) {
      vm->storage[secondary_idx2 * vm->PAGESIZE + tr2] = vm->buffer[swap_idx * vm->PAGESIZE + tr2];
    } 
  } else if (flag2 == 1 && flag3 == 0) {  // the page does not exist in the secondary memory
    vm->invert_page_table[swap_idx + vm->PAGE_ENTRIES] = page_num;  
  }

  vm->buffer[(swap_idx * vm->PAGESIZE) + (addr % vm->PAGESIZE)] = value;

  /* maintain the LRU info */
  vm->invert_page_table[3 * vm->PAGE_ENTRIES] += 1;
  vm->invert_page_table[2 * vm->PAGE_ENTRIES + swap_idx] = vm->invert_page_table[3 * vm->PAGE_ENTRIES];
  vm->invert_page_table[3 * vm->PAGE_ENTRIES + 1] = find_LRU(vm);

  return;
}

__device__ void vm_snapshot(VirtualMemory *vm, uchar *results, int offset,
                            int input_size) {
  /* Complete snapshot function togther with vm_read to load elements from data
   * to result buffer */
  for (int k = 0; k < input_size; k++) {
    results[k] = vm_read(vm, k + offset);
  }
}

